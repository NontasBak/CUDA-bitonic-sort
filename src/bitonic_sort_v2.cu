#include <assert.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__device__ void swap(int *arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}

__global__ void exchange(int *arr, int distance, int group_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int i = (tid / distance) * distance * 2 + (tid % distance);
    int partner = i ^ distance;

    if ((i & group_size) == 0 && arr[i] > arr[partner]) {
        swap(arr, i, partner);
    } else if ((i & group_size) != 0 && arr[i] < arr[partner]) {
        swap(arr, i, partner);
    }
}

__global__ void initialExchangeLocally(int *arr) {
    int t = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + t;

    int offset = blockIdx.x * blockDim.x * 2; // 0, 2048, 4096, ...

    // Transfer data to shared memory
    __shared__ int shared_arr[2048];
    shared_arr[t] = arr[offset + t];
    shared_arr[t + blockDim.x] = arr[offset + t + blockDim.x];
    __syncthreads();

    for (int group_size = 2; group_size <= 2048; group_size <<= 1) {
        for (int distance = group_size >> 1; distance > 0; distance >>= 1) {
            int i_global = (tid / distance) * distance * 2 + (tid % distance);
            int i = (t / distance) * distance * 2 + (t % distance);
            int partner = i ^ distance;

            if ((i_global & group_size) == 0 &&
                shared_arr[i] > shared_arr[partner]) {
                swap(shared_arr, i, partner);
            } else if ((i_global & group_size) != 0 &&
                       shared_arr[i] < shared_arr[partner]) {
                swap(shared_arr, i, partner);
            }
            __syncthreads();
        }
    }

    // Transfer data back to global memory
    arr[offset + t] = shared_arr[t];
    arr[offset + t + blockDim.x] = shared_arr[t + blockDim.x];
}

__global__ void exchangeLocally(int *arr, int group_size) {
    int t = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + t;

    int offset = blockIdx.x * blockDim.x * 2; // 0, 2048, 4096, ...

    // Transfer data to shared memory
    __shared__ int shared_arr[2048];
    shared_arr[t] = arr[offset + t];
    shared_arr[t + blockDim.x] = arr[offset + t + blockDim.x];
    __syncthreads();

    for (int distance = 1024; distance > 0; distance >>= 1) {
        int i_global = (tid / distance) * distance * 2 + (tid % distance);
        int i = (t / distance) * distance * 2 + (t % distance);
        int partner = i ^ distance;

        if ((i_global & group_size) == 0 &&
            shared_arr[i] > shared_arr[partner]) {
            swap(shared_arr, i, partner);
        } else if ((i_global & group_size) != 0 &&
                   shared_arr[i] < shared_arr[partner]) {
            swap(shared_arr, i, partner);
        }
        __syncthreads();
    }

    // Transfer data back to global memory
    arr[offset + t] = shared_arr[t];
    arr[offset + t + blockDim.x] = shared_arr[t + blockDim.x];
}

void bitonicSort(int *d_arr, int n, int num_threads, int num_blocks) {
    initialExchangeLocally<<<num_blocks, num_threads>>>(d_arr);

    for (int group_size = 4096; group_size <= n; group_size <<= 1) {
        for (int distance = group_size >> 1; distance > 1024; distance >>= 1) {
            exchange<<<num_blocks, num_threads>>>(d_arr, distance, group_size);
        }
        exchangeLocally<<<num_blocks, num_threads>>>(d_arr, group_size);
    }
}

int main(int argc, char *argv[]) {
    if (argc != 2) {
        printf("Usage: %s <power_of_2>\n", argv[0]);
        printf("Example: %s 27 for 2^27 elements\n", argv[0]);
        return 1;
    }

    int power = atoi(argv[1]);
    int n = 1 << power;
    int num_threads = 1 << 10;
    int num_blocks = n / (2 * num_threads);

    printf("Executing V2 with %d elements (2^%d)\n", n, power);

    int *arr = (int *)malloc(n * sizeof(int));
    int *d_arr;

    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 100;
    }

    struct timespec start_total, end_total, start, end;
    double malloc_time, h2d_time, sort_time, d2h_time, total_time,
        execution_time;

    // Start total timing
    clock_gettime(CLOCK_MONOTONIC, &start_total);

    // Measure cudaMalloc
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMalloc(&d_arr, n * sizeof(int));
    clock_gettime(CLOCK_MONOTONIC, &end);
    malloc_time =
        (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

    // Measure Host to Device copy
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    clock_gettime(CLOCK_MONOTONIC, &end);
    h2d_time =
        (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

    // Measure sort algorithm
    clock_gettime(CLOCK_MONOTONIC, &start);
    bitonicSort(d_arr, n, num_threads, num_blocks);
    clock_gettime(CLOCK_MONOTONIC, &end);
    sort_time =
        (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

    // Measure Device to Host copy
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(arr, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_MONOTONIC, &end);
    d2h_time =
        (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;

    clock_gettime(CLOCK_MONOTONIC, &end_total);
    total_time = (end_total.tv_sec - start_total.tv_sec) +
                 (end_total.tv_nsec - start_total.tv_nsec) / 1e9;
    execution_time = h2d_time + sort_time + d2h_time;

    printf("\nTiming Results:\n");
    printf("hipMalloc time: %f seconds\n", malloc_time);
    printf("Host to Device copy time: %f seconds\n", h2d_time);
    printf("Sort algorithm time: %f seconds\n", sort_time);
    printf("Device to Host copy time: %f seconds\n", d2h_time);
    printf("Total time (including malloc): %f seconds\n", total_time);
    printf("Execution time (excluding malloc): %f seconds\n", execution_time);

    // Verification
    for (int i = 1; i < n; i++) {
        assert(arr[i - 1] <= arr[i]);
    }
    printf("PASSED\n");

    hipFree(d_arr);
    free(arr);

    return 0;
}