#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>


__global__ void exchange(int *arr, int size, int distance, int group_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int partner = tid ^ distance;
    if(partner > tid) {
        if((tid & group_size) == 0 && arr[tid] > arr[partner]) {
            int temp = arr[tid];
            arr[tid] = arr[partner];
            arr[partner] = temp;
        }
        if((tid & group_size) != 0 && arr[tid] < arr[partner]) {
            int temp = arr[tid];
            arr[tid] = arr[partner];
            arr[partner] = temp;
        }
    }
}

int main() {
    int n = 2048;
    int num_threads = 1024;
    int num_blocks = n / num_threads;
    
    int *arr = (int *)malloc(n * sizeof(int));
    int *out = (int *)malloc(n * sizeof(int));
    int *d_arr;

    for(int i = 0; i < n; i++) {
        arr[i] = rand() % 100;
    }

    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    
    for(int group_size = 2; group_size <= n; group_size <<= 1) {
        for(int distance = group_size >> 1; distance > 0; distance >>= 1) {
            exchange<<<num_blocks, num_threads>>>(d_arr, n, distance, group_size);
        }
    }

    hipMemcpy(out, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 1; i < n; i++) {
        assert(out[i - 1] <= out[i]);
    }

    printf("PASSED\n");

    hipFree(d_arr);

    free(arr);
    free(out);

    return 0;
}