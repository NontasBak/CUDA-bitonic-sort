#include <assert.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

__device__ void swap(int *arr, int i, int j) {
    int temp = arr[i];
    arr[i] = arr[j];
    arr[j] = temp;
}

__global__ void exchange(int *arr, int size, int distance, int group_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int partner = tid ^ distance;
    if (partner > tid) {
        if ((tid & group_size) == 0 && arr[tid] > arr[partner]) {
            swap(arr, tid, partner);
        }
        if ((tid & group_size) != 0 && arr[tid] < arr[partner]) {
            swap(arr, tid, partner);
        }
    }
}

__global__ void initialExchangeLocally(int *arr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool sort_descending = blockIdx.x & 1;

    for (int group_size = 2; group_size <= size; group_size <<= 1) {
        for (int distance = group_size >> 1; distance > 0; distance >>= 1) {
            int partner = tid ^ distance;
            if (partner > tid) {
                if (group_size == 1024 && sort_descending) {
                    if (arr[tid] < arr[partner]) {
                        swap(arr, tid, partner);
                    }
                } else {
                    if ((tid & group_size) == 0 && arr[tid] > arr[partner]) {
                        swap(arr, tid, partner);
                    }
                    if ((tid & group_size) != 0 && arr[tid] < arr[partner]) {
                        swap(arr, tid, partner);
                    }
                }
            }
            __syncthreads();
        }
    }
}

__global__ void exchangeLocally(int *arr, int group_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int distance = 1024 >> 1; distance > 0; distance >>= 1) {
        int partner = tid ^ distance;
        if (partner > tid) {
            if ((tid & group_size) == 0 && arr[tid] > arr[partner]) {
                swap(arr, tid, partner);
            }
            if ((tid & group_size) != 0 && arr[tid] < arr[partner]) {
                swap(arr, tid, partner);
            }
        }
        __syncthreads();
    }
}

int main() {
    int n = 1 << 13;
    int num_threads = 1 << 10;
    int num_blocks = n / num_threads;

    int *arr = (int *)malloc(n * sizeof(int));
    int *out = (int *)malloc(n * sizeof(int));
    int *d_arr;

    for (int i = 0; i < n; i++) {
        arr[i] = rand() % 100;
    }

    hipMalloc(&d_arr, n * sizeof(int));
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    initialExchangeLocally<<<num_blocks, num_threads>>>(d_arr, 1 << 10);

    for (int group_size = 2048; group_size <= n; group_size <<= 1) {
        for (int distance = group_size >> 1; distance > 512; distance >>= 1) {
            exchange<<<num_blocks, num_threads>>>(d_arr, n, distance,
                                                  group_size);
        }
        exchangeLocally<<<num_blocks, num_threads>>>(d_arr, group_size);
    }

    hipMemcpy(out, d_arr, n * sizeof(int), hipMemcpyDeviceToHost);

    // for (int i = 0; i < n; i++) {
    //     if (i % 1024 == 0) printf("\n\n");
    //     printf("%d ", out[i]);
    // }

    for (int i = 1; i < n; i++) {
        assert(out[i - 1] <= out[i]);
    }

    printf("PASSED\n");

    hipFree(d_arr);

    free(arr);
    free(out);

    return 0;
}